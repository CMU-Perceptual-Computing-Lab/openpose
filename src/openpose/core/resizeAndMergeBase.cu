#include "hip/hip_runtime.h"
#include <openpose/utilities/cuda.hpp>
#include <openpose/utilities/cuda.hu>
#include <openpose/core/resizeAndMergeBase.hpp>

namespace op
{
    const auto THREADS_PER_BLOCK_1D = 16u;

    template <typename T>
    __global__ void resizeKernel(T* targetPtr, const T* const sourcePtr, const int sourceWidth, const int sourceHeight,
                                 const int targetWidth, const int targetHeight)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;

        if (x < targetWidth && y < targetHeight)
        {
            const T xSource = (x + T(0.5f)) * sourceWidth / T(targetWidth) - T(0.5f);
            const T ySource = (y + T(0.5f)) * sourceHeight / T(targetHeight) - T(0.5f);
            targetPtr[y*targetWidth+x] = bicubicInterpolate(sourcePtr, xSource, ySource, sourceWidth, sourceHeight,
                                                            sourceWidth);
        }
    }

    template <typename T>
    __global__ void resizeKernelAndAdd(T* targetPtr, const T* const sourcePtr, const T scaleWidth,
                                       const T scaleHeight, const int sourceWidth, const int sourceHeight,
                                       const int targetWidth, const int targetHeight)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;

        if (x < targetWidth && y < targetHeight)
        {
            const T xSource = (x + T(0.5f)) / scaleWidth - T(0.5f);
            const T ySource = (y + T(0.5f)) / scaleHeight - T(0.5f);
            targetPtr[y*targetWidth+x] += bicubicInterpolate(sourcePtr, xSource, ySource, sourceWidth, sourceHeight,
                                                             sourceWidth);
        }
    }

    template <typename T>
    __global__ void resizeKernelAndAverage(T* targetPtr, const T* const sourcePtr, const T scaleWidth,
                                           const T scaleHeight, const int sourceWidth, const int sourceHeight,
                                           const int targetWidth, const int targetHeight, const int counter)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;

        if (x < targetWidth && y < targetHeight)
        {
            const T xSource = (x + T(0.5f)) / scaleWidth - T(0.5f);
            const T ySource = (y + T(0.5f)) / scaleHeight - T(0.5f);
            const auto interpolated = bicubicInterpolate(sourcePtr, xSource, ySource, sourceWidth, sourceHeight,
                                                         sourceWidth);
            auto& targetPixel = targetPtr[y*targetWidth+x];
            targetPixel = (targetPixel + interpolated) / T(counter);
        }
    }

    template <typename T>
    void resizeAndMergeGpu(T* targetPtr, const std::vector<const T*>& sourcePtrs, const std::array<int, 4>& targetSize,
                           const std::vector<std::array<int, 4>>& sourceSizes,
                           const std::vector<T>& scaleInputToNetInputs)
    {
        try
        {
            // Security checks
            if (sourceSizes.empty())
                error("sourceSizes cannot be empty.", __LINE__, __FUNCTION__, __FILE__);
            if (sourcePtrs.size() != sourceSizes.size() || sourceSizes.size() != scaleInputToNetInputs.size())
                error("Size(sourcePtrs) must match size(sourceSizes) and size(scaleInputToNetInputs). Currently: "
                      + std::to_string(sourcePtrs.size()) + " vs. " + std::to_string(sourceSizes.size()) + " vs. "
                      + std::to_string(scaleInputToNetInputs.size()) + ".", __LINE__, __FUNCTION__, __FILE__);

            // Parameters
            const auto channels = targetSize[1];
            const auto targetHeight = targetSize[2];
            const auto targetWidth = targetSize[3];
            const dim3 threadsPerBlock{THREADS_PER_BLOCK_1D, THREADS_PER_BLOCK_1D};
            const dim3 numBlocks{getNumberCudaBlocks(targetWidth, threadsPerBlock.x),
                                 getNumberCudaBlocks(targetHeight, threadsPerBlock.y)};
            const auto& sourceSize = sourceSizes[0];
            const auto sourceHeight = sourceSize[2];
            const auto sourceWidth = sourceSize[3];

            // No multi-scale merging or no merging required
            if (sourceSizes.size() == 1)
            {
                const auto num = sourceSize[0];
                if (targetSize[0] > 1 || num == 1)
                {
                    const auto sourceChannelOffset = sourceHeight * sourceWidth;
                    const auto targetChannelOffset = targetWidth * targetHeight;
                    for (auto n = 0; n < num; n++)
                    {
                        const auto offsetBase = n*channels;
                        for (auto c = 0 ; c < channels ; c++)
                        {
                            const auto offset = offsetBase + c;
                            resizeKernel<<<numBlocks, threadsPerBlock>>>(targetPtr + offset * targetChannelOffset,
                                                                         sourcePtrs.at(0) + offset * sourceChannelOffset,
                                                                         sourceWidth, sourceHeight, targetWidth,
                                                                         targetHeight);
                        }
                    }
                }
                // Old inefficient multi-scale merging
                else
                    error("It should never reache this point. Notify us otherwise.", __LINE__, __FUNCTION__, __FILE__);
            }
            // Multi-scaling merging
            else
            {
                const auto targetChannelOffset = targetWidth * targetHeight;
                hipMemset(targetPtr, 0.f, channels*targetChannelOffset * sizeof(T));
                const auto scaleToMainScaleWidth = targetWidth / T(sourceWidth);
                const auto scaleToMainScaleHeight = targetHeight / T(sourceHeight);

                for (auto i = 0u ; i < sourceSizes.size(); i++)
                {
                    const auto& currentSize = sourceSizes.at(i);
                    const auto currentHeight = currentSize[2];
                    const auto currentWidth = currentSize[3];
                    const auto sourceChannelOffset = currentHeight * currentWidth;
                    const auto scaleInputToNet = scaleInputToNetInputs[i] / scaleInputToNetInputs[0];
                    const auto scaleWidth = scaleToMainScaleWidth / scaleInputToNet;
                    const auto scaleHeight = scaleToMainScaleHeight / scaleInputToNet;
                    // All but last image --> add
                    if (i < sourceSizes.size() - 1)
                    {
                        for (auto c = 0 ; c < channels ; c++)
                        {
                            resizeKernelAndAdd<<<numBlocks, threadsPerBlock>>>(
                                targetPtr + c * targetChannelOffset, sourcePtrs[i] + c * sourceChannelOffset,
                                scaleWidth, scaleHeight, currentWidth, currentHeight, targetWidth,
                                targetHeight
                            );
                        }
                    }
                    // Last image --> average all
                    else
                    {
                        for (auto c = 0 ; c < channels ; c++)
                        {
                            resizeKernelAndAverage<<<numBlocks, threadsPerBlock>>>(
                                targetPtr + c * targetChannelOffset, sourcePtrs[i] + c * sourceChannelOffset,
                                scaleWidth, scaleHeight, currentWidth, currentHeight, targetWidth,
                                targetHeight, sourceSizes.size()
                            );
                        }
                    }
                }
            }

            cudaCheck(__LINE__, __FUNCTION__, __FILE__);
        }
        catch (const std::exception& e)
        {
            error(e.what(), __LINE__, __FUNCTION__, __FILE__);
        }
    }

    template void resizeAndMergeGpu(float* targetPtr, const std::vector<const float*>& sourcePtrs,
                                    const std::array<int, 4>& targetSize,
                                    const std::vector<std::array<int, 4>>& sourceSizes,
                                    const std::vector<float>& scaleInputToNetInputs);
    template void resizeAndMergeGpu(double* targetPtr, const std::vector<const double*>& sourcePtrs,
                                    const std::array<int, 4>& targetSize,
                                    const std::vector<std::array<int, 4>>& sourceSizes,
                                    const std::vector<double>& scaleInputToNetInputs);
}
