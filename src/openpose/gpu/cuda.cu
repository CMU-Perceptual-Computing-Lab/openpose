#include "hip/hip_runtime.h"
#ifdef USE_CUDA
    #include <hip/hip_runtime.h>
    #include <hip/hip_runtime.h>
    #include <openpose/gpu/cuda.hu>
#endif
#include <openpose/gpu/cuda.hpp>

namespace op
{
    template <typename T>
    __global__ void reorderAndNormalizeKernel(
        T* targetPtr, const unsigned char* const srcPtr, const int width, const int height, const int channels)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
        const auto c = (blockIdx.z * blockDim.z) + threadIdx.z;
        if (x < width && y < height)
        {
            const auto originFramePtrOffsetY = y * width;
            const auto channelOffset = c * width * height;
            const auto targetIndex = channelOffset + y * width + x;
            const auto srcIndex = (originFramePtrOffsetY + x) * channels + c;
            targetPtr[targetIndex] =  T(srcPtr[srcIndex]) * T(1/256.f) - T(0.5f);
        }
    }

    template <typename T>
    __global__ void uCharImageCastKernel(
        unsigned char* targetPtr, const T* const srcPtr, const int volume)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (x < volume)
            targetPtr[x] =  (unsigned char)(fastTruncateCuda(srcPtr[x], T(0), T(255)));
    }

    template <typename T>
    void reorderAndNormalize(
        T* targetPtr, const unsigned char* const srcPtr, const int width, const int height, const int channels)
    {
        try
        {
            const dim3 threadsPerBlock{32, 1, 1};
            const dim3 numBlocks{
                getNumberCudaBlocks(width, threadsPerBlock.x),
                getNumberCudaBlocks(height, threadsPerBlock.y),
                getNumberCudaBlocks(channels, threadsPerBlock.z)};
            reorderAndNormalizeKernel<<<numBlocks, threadsPerBlock>>>(targetPtr, srcPtr, width, height, channels);
        }
        catch (const std::exception& e)
        {
            error(e.what(), __LINE__, __FUNCTION__, __FILE__);
        }
    }

    template <typename T>
    void uCharImageCast(unsigned char* targetPtr, const T* const srcPtr, const int volume)
    {
        try
        {
            const dim3 threadsPerBlock{32, 1, 1};
            const dim3 numBlocks{
                getNumberCudaBlocks(volume, threadsPerBlock.x)};
            uCharImageCastKernel<<<numBlocks, threadsPerBlock>>>(targetPtr, srcPtr, volume);
        }
        catch (const std::exception& e)
        {
            error(e.what(), __LINE__, __FUNCTION__, __FILE__);
        }
    }

    template void reorderAndNormalize(
        float* targetPtr, const unsigned char* const srcPtr, const int width, const int height, const int channels);
    template void reorderAndNormalize(
        double* targetPtr, const unsigned char* const srcPtr, const int width, const int height, const int channels);

    template void uCharImageCast(
        unsigned char* targetPtr, const float* const srcPtr, const int volume);
    template void uCharImageCast(
        unsigned char* targetPtr, const double* const srcPtr, const int volume);
}
