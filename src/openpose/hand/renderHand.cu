#include "hip/hip_runtime.h"
#include <openpose/hand/handParameters.hpp>
#include <openpose/utilities/errorAndLog.hpp>
#include <openpose/utilities/cuda.hpp>
#include <openpose/utilities/cuda.hu>
#include <openpose/utilities/render.hu>
#include <openpose/hand/renderHand.hpp>

namespace op
{
    __constant__ const unsigned int PART_PAIRS_GPU[] = HAND_PAIRS_RENDER_GPU;
    __constant__ const float COLORS[] = {HAND_COLORS_RENDER};



    __global__ void renderHandsParts(float* targetPtr, const int targetWidth, const int targetHeight,
                                     const float* const handsPtr, const int numberHands,
                                     const float threshold, const float alphaColorToAdd)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
        const auto globalIdx = threadIdx.y * blockDim.x + threadIdx.x;

        // Shared parameters
        __shared__ float2 sharedMins[HAND_MAX_HANDS];
        __shared__ float2 sharedMaxs[HAND_MAX_HANDS];
        __shared__ float sharedScaleF[HAND_MAX_HANDS];

        // Other parameters
        const auto numberPartPairs = sizeof(PART_PAIRS_GPU) / (2*sizeof(PART_PAIRS_GPU[0]));
        const auto numberColors = sizeof(COLORS) / (3*sizeof(COLORS[0]));
        const auto radius = fastMin(targetWidth, targetHeight) / 100.f;
        const auto stickwidth = fastMin(targetWidth, targetHeight) / 80.f;

        // Render key points
        renderKeypoints(targetPtr, sharedMaxs, sharedMins, sharedScaleF,
                        globalIdx, x, y, targetWidth, targetHeight, handsPtr, PART_PAIRS_GPU, numberHands,
                        HAND_NUMBER_PARTS, numberPartPairs, COLORS, numberColors,
                        radius, stickwidth, threshold, alphaColorToAdd);
    }

    void renderHandKeypointsGpu(float* framePtr, const Point<int>& frameSize, const float* const handsPtr, const int numberHands,
                                const float alphaColorToAdd)
    {
        try
        {
            if (numberHands > 0)
            {
                dim3 threadsPerBlock;
                dim3 numBlocks;
                std::tie(threadsPerBlock, numBlocks) = getNumberCudaThreadsAndBlocks(frameSize);
                renderHandsParts<<<threadsPerBlock, numBlocks>>>(framePtr, frameSize.x, frameSize.y, handsPtr,
                                                                 numberHands, HAND_RENDER_THRESHOLD, alphaColorToAdd);
                cudaCheck(__LINE__, __FUNCTION__, __FILE__);
            }
        }
        catch (const std::exception& e)
        {
            error(e.what(), __LINE__, __FUNCTION__, __FILE__);
        }
    }
}
