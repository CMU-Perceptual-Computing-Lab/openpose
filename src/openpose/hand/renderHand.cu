#include "hip/hip_runtime.h"
#include <openpose/gpu/cuda.hpp>
#include <openpose/gpu/cuda.hu>
#include <openpose/hand/handParameters.hpp>
#include <openpose/utilities/render.hu>
#include <openpose/hand/renderHand.hpp>

namespace op
{
    __constant__ const unsigned int PART_PAIRS_GPU[] = {HAND_PAIRS_RENDER_GPU};
    __constant__ const float SCALES[] = {HAND_SCALES_RENDER_GPU};
    __constant__ const float COLORS[] = {HAND_COLORS_RENDER_GPU};

    __global__ void renderHandsParts(float* targetPtr, const int targetWidth, const int targetHeight,
                                     const float* const handsPtr, const int numberHands,
                                     const float threshold, const float alphaColorToAdd)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
        const auto globalIdx = threadIdx.y * blockDim.x + threadIdx.x;

        // Shared parameters
        __shared__ float2 sharedMins[HAND_MAX_HANDS];
        __shared__ float2 sharedMaxs[HAND_MAX_HANDS];
        __shared__ float sharedScaleF[HAND_MAX_HANDS];

        // Other parameters
        const auto numberPartPairs = sizeof(PART_PAIRS_GPU) / (2*sizeof(PART_PAIRS_GPU[0]));
        const auto numberScales = sizeof(SCALES) / sizeof(SCALES[0]);
        const auto numberColors = sizeof(COLORS) / (3*sizeof(COLORS[0]));
        const auto radius = fastMin(targetWidth, targetHeight) / 100.f;
        const auto lineWidth = fastMin(targetWidth, targetHeight) / 80.f;

        // Render key points
        renderKeypoints(targetPtr, sharedMaxs, sharedMins, sharedScaleF, globalIdx, x, y, targetWidth, targetHeight,
                        handsPtr, PART_PAIRS_GPU, numberHands, HAND_NUMBER_PARTS, numberPartPairs, COLORS,
                        numberColors, radius, lineWidth, SCALES, numberScales, threshold, alphaColorToAdd);
    }

    void renderHandKeypointsGpu(float* framePtr, const Point<int>& frameSize, const float* const handsPtr,
                                const int numberHands, const float renderThreshold, const float alphaColorToAdd)
    {
        try
        {
            if (numberHands > 0)
            {
                dim3 threadsPerBlock;
                dim3 numBlocks;
                getNumberCudaThreadsAndBlocks(threadsPerBlock, numBlocks, frameSize);
                renderHandsParts<<<threadsPerBlock, numBlocks>>>(framePtr, frameSize.x, frameSize.y, handsPtr,
                                                                 numberHands, renderThreshold, alphaColorToAdd);
                cudaCheck(__LINE__, __FUNCTION__, __FILE__);
            }
        }
        catch (const std::exception& e)
        {
            error(e.what(), __LINE__, __FUNCTION__, __FILE__);
        }
    }
}
