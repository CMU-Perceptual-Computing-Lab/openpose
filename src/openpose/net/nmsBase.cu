#include "hip/hip_runtime.h"
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <openpose/gpu/cuda.hpp>
#include <openpose/gpu/cuda.hu>
#include <openpose/net/nmsBase.hpp>

namespace op
{
    const auto THREADS_PER_BLOCK_1D = 16u;
    const auto THREADS_PER_BLOCK = 512u;

    template <typename T>
    __global__ void nmsRegisterKernel(int* kernelPtr, const T* const sourcePtr, const int w, const int h,
                                      const T threshold)
    {
        // get pixel location (x,y)
        const auto x = blockIdx.x * blockDim.x + threadIdx.x;
        const auto y = blockIdx.y * blockDim.y + threadIdx.y;
        const auto index = y*w + x;

        if (0 < x && x < (w-1) && 0 < y && y < (h-1))
        {
            const auto value = sourcePtr[index];
            if (value > threshold)
            {
                const auto topLeft     = sourcePtr[(y-1)*w + x-1];
                const auto top         = sourcePtr[(y-1)*w + x];
                const auto topRight    = sourcePtr[(y-1)*w + x+1];
                const auto left        = sourcePtr[    y*w + x-1];
                const auto right       = sourcePtr[    y*w + x+1];
                const auto bottomLeft  = sourcePtr[(y+1)*w + x-1];
                const auto bottom      = sourcePtr[(y+1)*w + x];
                const auto bottomRight = sourcePtr[(y+1)*w + x+1];

                if (value > topLeft && value > top && value > topRight
                    && value > left && value > right
                    && value > bottomLeft && value > bottom && value > bottomRight)
                    kernelPtr[index] = 1;
                else
                    kernelPtr[index] = 0;
            }
            else
                kernelPtr[index] = 0;
        }
        else if (x == 0 || x == (w-1) || y == 0 || y == (h-1))
            kernelPtr[index] = 0;
    }

    template <typename T>
    __global__ void writeResultKernel(T* output, const int length, const int* const kernelPtr,
                                      const T* const sourcePtr, const int width, const int height, const int maxPeaks,
                                      const T offsetX, const T offsetY)
    {
        __shared__ int local[THREADS_PER_BLOCK+1]; // one more
        const auto globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

        if (globalIdx < length)
        {
            local[threadIdx.x] = kernelPtr[globalIdx];
            //last thread in the block but not globally last, load one more
            if (threadIdx.x == THREADS_PER_BLOCK - 1 && globalIdx != length - 1)
                local[threadIdx.x+1] = kernelPtr[globalIdx+1];

            __syncthreads();
            // See difference, except the globally last one
            if (globalIdx != length - 1)
            {
                // A[globalIdx] == A[globalIdx + 1] means no peak
                if (local[threadIdx.x] != local[threadIdx.x + 1])
                {
                    const auto peakIndex = kernelPtr[globalIdx]; //0-index
                    const auto peakLocX = (int)(globalIdx % width);
                    const auto peakLocY = (int)(globalIdx / width);

                    // Accurate peak location: considered neighboors
                    if (peakIndex < maxPeaks) // limitation
                    {
                        T xAcc = 0.f;
                        T yAcc = 0.f;
                        T scoreAcc = 0.f;
                        const auto dWidth = 3;
                        const auto dHeight = 3;
                        for (auto dy = -dHeight ; dy <= dHeight ; dy++)
                        {
                            const auto y = peakLocY + dy;
                            if (0 <= y && y < height) // Default height = 368
                            {
                                for (auto dx = -dWidth ; dx <= dWidth ; dx++)
                                {
                                    const auto x = peakLocX + dx;
                                    if (0 <= x && x < width) // Default width = 656
                                    {
                                        const auto score = sourcePtr[y * width + x];
                                        if (score > 0)
                                        {
                                            xAcc += x*score;
                                            yAcc += y*score;
                                            scoreAcc += score;
                                        }
                                    }
                                }
                            }
                        }

                        // Offset to keep Matlab format (empirically higher acc)
                        // Best results for 1 scale: x + 0, y + 0.5
                        // +0.5 to both to keep Matlab format
                        const auto outputIndex = (peakIndex + 1) * 3;
                        output[outputIndex] = xAcc / scoreAcc + offsetX;
                        output[outputIndex + 1] = yAcc / scoreAcc + offsetY;
                        output[outputIndex + 2] = sourcePtr[peakLocY*width + peakLocX];
                    }
                }
            }
            // If index 0 --> Assign number of peaks (truncated to the maximum possible number of peaks)
            else
                output[0] = (kernelPtr[globalIdx] < maxPeaks ? kernelPtr[globalIdx] : maxPeaks);
        }
    }

    // template <typename T>
    // __global__ void sortKernel(T* targetPtr, const int channels, const int offsetTarget)
    // {
    //     const auto globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

    //     if (globalIdx < channels)
    //     {
    //         const auto totalOffset = globalIdx * offsetTarget;
    //         const int nonZeroElementsPlus1 = targetPtr[totalOffset]+1;
    //         for (auto i = 1 ; i < nonZeroElementsPlus1 ; i++)
    //         {
    //             // Find new maximum
    //             const auto iIndex = totalOffset+3*i;
    //             int maxIndex = i;
    //             T maxIndexValue = targetPtr[iIndex+2];
    //             for (auto j = i+1 ; j < nonZeroElementsPlus1 ; j++)
    //             {
    //                 if (maxIndexValue < targetPtr[totalOffset+3*j+2])
    //                 {
    //                     maxIndex = j;
    //                     maxIndexValue = targetPtr[totalOffset+3*j+2];
    //                 }
    //             }
    //             // Swap
    //             const auto jIndex = totalOffset+3*maxIndex;
    //             const T temp [3] = {targetPtr[iIndex],
    //                                 targetPtr[iIndex+1],
    //                                 targetPtr[iIndex+2]};
    //             targetPtr[iIndex] = targetPtr[jIndex];
    //             targetPtr[iIndex+1] = targetPtr[jIndex+1];
    //             targetPtr[iIndex+2] = targetPtr[jIndex+2];
    //             targetPtr[jIndex] = temp[0];
    //             targetPtr[jIndex+1] = temp[1];
    //             targetPtr[jIndex+2] = temp[2];
    //         }
    //     }
    // }

    template <typename T>
    void nmsGpu(T* targetPtr, int* kernelPtr, const T* const sourcePtr, const T threshold,
                const std::array<int, 4>& targetSize, const std::array<int, 4>& sourceSize, const Point<T>& offset)
    {
        try
        {
            //Forward_cpu(bottom, top);
            const auto num = sourceSize[0];
            const auto height = sourceSize[2];
            const auto width = sourceSize[3];
            const auto channels = targetSize[1];
            const auto maxPeaks = targetSize[2]-1;
            const auto imageOffset = height * width;
            const auto offsetTarget = (maxPeaks+1)*targetSize[3];

            const dim3 threadsPerBlock2D{THREADS_PER_BLOCK_1D, THREADS_PER_BLOCK_1D};
            const dim3 numBlocks2D{getNumberCudaBlocks(width, threadsPerBlock2D.x),
                                   getNumberCudaBlocks(height, threadsPerBlock2D.y)};
            const dim3 threadsPerBlock1D{THREADS_PER_BLOCK};
            const dim3 numBlocks1D{getNumberCudaBlocks(imageOffset, threadsPerBlock1D.x)};
            // const dim3 threadsPerBlockSort{128};
            // const dim3 numBlocksSort{getNumberCudaBlocks(channels, threadsPerBlockSort.x)};
            // log("num_b: " + std::to_string(sourceSize[0]));       // = 1
            // log("channel_b: " + std::to_string(sourceSize[1]));   // = 57 = 18 body parts + bkg + 19x2 PAFs
            // log("height_b: " + std::to_string(sourceSize[2]));    // = 368 = height
            // log("width_b: " + std::to_string(sourceSize[3]));     // = 656 = width
            // log("num_t: " + std::to_string(targetSize[0]));       // = 1
            // log("channel_t: " + std::to_string(targetSize[1]));   // = 18 = numberParts
            // log("height_t: " + std::to_string(targetSize[2]));    // = 128 = maxPeople + 1
            // log("width_t: " + std::to_string(targetSize[3]));     // = 3 = [x, y, score]
            // log("");

            for (auto n = 0; n < num; n++)
            {
                for (auto c = 0; c < channels; c++)
                {
                    // log("channel: " + std::to_string(c));
                    const auto offsetChannel = (n * channels + c);
                    auto* kernelPtrOffsetted = kernelPtr + offsetChannel * imageOffset;
                    const auto* const sourcePtrOffsetted = sourcePtr + offsetChannel * imageOffset;
                    auto* targetPtrOffsetted = targetPtr + offsetChannel * offsetTarget;

                    // This returns kernelPtrOffsetted, a binary array with 0s & 1s. 1s in the local maximum
                    // positions (size = size(sourcePtrOffsetted))
                    // Example result: [0,0,0,0,1,0,0,0,0,1,0,0,0,0]
                    nmsRegisterKernel<<<numBlocks2D, threadsPerBlock2D>>>(
                        kernelPtrOffsetted, sourcePtrOffsetted, width, height, threshold);
                    // // Debug
                    // if (c==3)
                    // {
                    //     char filename[50];
                    //     sprintf(filename, "work%02d.txt", c);
                    //     std::ofstream fout(filename);
                    //     int* kernelPtrOffsetted_local = mKernelBlob.mutable_cpu_data()
                    //                                   + n * parts_num * imageOffset + c * imageOffset;
                    //     for (int y = 0; y < height; y++){
                    //         for (int x = 0; x < width; x++)
                    //             fout << kernelPtrOffsetted_local[y*width + x] << "\t";
                    //         fout<< std::endl;
                    //     }
                    //     fout.close();
                    // }
                    auto kernelThrustPtr = thrust::device_pointer_cast(kernelPtrOffsetted);

                    // This modifies kernelPtrOffsetted, now it indicates the local maximum indexes
                    // Format: 0,0,0,1,1,1,1,2,2,2,... First maximum at index 2, second at 6, etc...
                    // Example result: [0,0,0,0,0,1,1,1,1,1,2,2,2,2]
                    thrust::exclusive_scan(kernelThrustPtr, kernelThrustPtr + imageOffset, kernelThrustPtr);

                    // This returns targetPtrOffsetted, with the NMS applied over it
                    writeResultKernel<<<numBlocks1D, threadsPerBlock1D>>>(targetPtrOffsetted, imageOffset,
                                                                          kernelPtrOffsetted, sourcePtrOffsetted,
                                                                          width, height, maxPeaks, offset.x, offset.y);

                }
                // // Sort based on score
                // // Commented because it doesn't change accuracy
                // // TODO: If finally used, implement for CPU/CL versions
                // sortKernel<<<numBlocksSort, threadsPerBlockSort>>>(targetPtr, channels, offsetTarget);
            }
            cudaCheck(__LINE__, __FUNCTION__, __FILE__);
        }
        catch (const std::exception& e)
        {
            error(e.what(), __LINE__, __FUNCTION__, __FILE__);
        }
    }

    template void nmsGpu(
        float* targetPtr, int* kernelPtr, const float* const sourcePtr, const float threshold,
        const std::array<int, 4>& targetSize, const std::array<int, 4>& sourceSize, const Point<float>& offset);
    template void nmsGpu(
        double* targetPtr, int* kernelPtr, const double* const sourcePtr, const double threshold,
        const std::array<int, 4>& targetSize, const std::array<int, 4>& sourceSize, const Point<double>& offset);
}
