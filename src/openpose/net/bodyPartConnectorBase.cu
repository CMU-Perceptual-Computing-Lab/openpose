#include "hip/hip_runtime.h"
#include <openpose/gpu/cuda.hpp>
#include <openpose/pose/poseParameters.hpp>
#include <openpose/utilities/fastMath.hpp>
#include <openpose/net/bodyPartConnectorBase.hpp>

namespace op
{
    const dim3 THREADS_PER_BLOCK{4, 16, 16};

    template<typename T>
    inline __device__ int intRoundGPU(const T a)
    {
        return int(a+T(0.5));
    }

    template <typename T>
    inline __device__  T process(const T* bodyPartA, const T* bodyPartB, const T* mapX, const T* mapY,
                                 const int heatmapWidth, const int heatmapHeight, const T interThreshold,
                                 const T interMinAboveThreshold)
    {
        const auto vectorAToBX = bodyPartB[0] - bodyPartA[0];
        const auto vectorAToBY = bodyPartB[1] - bodyPartA[1];
        const auto vectorAToBMax = max(abs(vectorAToBX), abs(vectorAToBY));
        const auto numberPointsInLine = max(5, min(25, intRoundGPU(sqrt(5*vectorAToBMax))));
        const auto vectorNorm = T(sqrt(vectorAToBX*vectorAToBX + vectorAToBY*vectorAToBY));

        if (vectorNorm > 1e-6)
        {
            const auto sX = bodyPartA[0];
            const auto sY = bodyPartA[1];
            const auto vectorAToBNormX = vectorAToBX/vectorNorm;
            const auto vectorAToBNormY = vectorAToBY/vectorNorm;

            auto sum = T(0.);
            auto count = 0;
            const auto vectorAToBXInLine = vectorAToBX/numberPointsInLine;
            const auto vectorAToBYInLine = vectorAToBY/numberPointsInLine;
            for (auto lm = 0; lm < numberPointsInLine; lm++)
            {
                const auto mX = min(heatmapWidth-1, intRoundGPU(sX + lm*vectorAToBXInLine));
                const auto mY = min(heatmapHeight-1, intRoundGPU(sY + lm*vectorAToBYInLine));
                const auto idx = mY * heatmapWidth + mX;
                const auto score = (vectorAToBNormX*mapX[idx] + vectorAToBNormY*mapY[idx]);
                if (score > interThreshold)
                {
                    sum += score;
                    count++;
                }
            }

            // Return PAF score
            if (count/T(numberPointsInLine) > interMinAboveThreshold)
                return sum/count;
            else
            {
                // Ideally, if distanceAB = 0, PAF is 0 between A and B, provoking a false negative
                // To fix it, we consider PAF-connected keypoints very close to have a minimum PAF score, such that:
                //     1. It will consider very close keypoints (where the PAF is 0)
                //     2. But it will not automatically connect them (case PAF score = 1), or real PAF might got
                //        missing
                const auto l2Dist = sqrtf(vectorAToBX*vectorAToBX + vectorAToBY*vectorAToBY);
                const auto threshold = sqrtf(heatmapWidth*heatmapHeight)/150; // 3.3 for 368x656, 6.6 for 2x resolution
                if (l2Dist < threshold)
                    return T(0.15);
            }
        }
        return -1;
    }

    template <typename T>
    __global__ void pafScoreKernel(T* pairScoresPtr, const T* const heatMapPtr, const T* const peaksPtr,
                                   const unsigned int* const bodyPartPairsPtr, const unsigned int* const mapIdxPtr,
                                   const unsigned int maxPeaks, const int numberBodyPartPairs,
                                   const int heatmapWidth, const int heatmapHeight, const T interThreshold,
                                   const T interMinAboveThreshold)
    {
        const auto pairIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto peakA = (blockIdx.y * blockDim.y) + threadIdx.y;
        const auto peakB = (blockIdx.z * blockDim.z) + threadIdx.z;

        if (pairIndex < numberBodyPartPairs && peakA < maxPeaks && peakB < maxPeaks)
        {
            const auto baseIndex = 2*pairIndex;
            const auto partA = bodyPartPairsPtr[baseIndex];
            const auto partB = bodyPartPairsPtr[baseIndex + 1];

            const T numberPeaksA = peaksPtr[3*partA*(maxPeaks+1)];
            const T numberPeaksB = peaksPtr[3*partB*(maxPeaks+1)];

            const auto outputIndex = (pairIndex*maxPeaks+peakA)*maxPeaks + peakB;
            if (peakA < numberPeaksA && peakB < numberPeaksB)
            {
                const auto mapIdxX = mapIdxPtr[baseIndex];
                const auto mapIdxY = mapIdxPtr[baseIndex + 1];

                const T* const bodyPartA = peaksPtr + (3*(partA*(maxPeaks+1) + peakA+1));
                const T* const bodyPartB = peaksPtr + (3*(partB*(maxPeaks+1) + peakB+1));
                const T* const mapX = heatMapPtr + mapIdxX*heatmapWidth*heatmapHeight;
                const T* const mapY = heatMapPtr + mapIdxY*heatmapWidth*heatmapHeight;
                pairScoresPtr[outputIndex] = process(
                    bodyPartA, bodyPartB, mapX, mapY, heatmapWidth, heatmapHeight, interThreshold,
                    interMinAboveThreshold);
            }
            else
                pairScoresPtr[outputIndex] = -1;
        }
    }

    template <typename T>
    void connectBodyPartsGpu(Array<T>& poseKeypoints, Array<T>& poseScores, const T* const heatMapGpuPtr,
                             const T* const peaksPtr, const PoseModel poseModel, const Point<int>& heatMapSize,
                             const int maxPeaks, const T interMinAboveThreshold, const T interThreshold,
                             const int minSubsetCnt, const T minSubsetScore, const T scaleFactor,
                             const bool maximizePositives, Array<T> pairScoresCpu, T* pairScoresGpuPtr,
                             const unsigned int* const bodyPartPairsGpuPtr, const unsigned int* const mapIdxGpuPtr,
                             const T* const peaksGpuPtr)
    {
        try
        {
            // Parts Connection
            const auto& bodyPartPairs = getPosePartPairs(poseModel);
            const auto numberBodyParts = getPoseNumberBodyParts(poseModel);
            const auto numberBodyPartPairs = (unsigned int)(bodyPartPairs.size() / 2);
            const auto totalComputations = pairScoresCpu.getVolume();

            if (numberBodyParts == 0)
                error("Invalid value of numberBodyParts, it must be positive, not " + std::to_string(numberBodyParts),
                      __LINE__, __FUNCTION__, __FILE__);
            if (bodyPartPairsGpuPtr == nullptr || mapIdxGpuPtr == nullptr)
                error("The pointers bodyPartPairsGpuPtr and mapIdxGpuPtr cannot be nullptr.",
                      __LINE__, __FUNCTION__, __FILE__);

            // Run Kernel - pairScoresGpu
            const dim3 numBlocks{
                getNumberCudaBlocks(numberBodyPartPairs, THREADS_PER_BLOCK.x),
                getNumberCudaBlocks(maxPeaks, THREADS_PER_BLOCK.y),
                getNumberCudaBlocks(maxPeaks, THREADS_PER_BLOCK.z)};
            pafScoreKernel<<<numBlocks, THREADS_PER_BLOCK>>>(
                pairScoresGpuPtr, heatMapGpuPtr, peaksGpuPtr, bodyPartPairsGpuPtr, mapIdxGpuPtr,
                maxPeaks, (int)numberBodyPartPairs, heatMapSize.x, heatMapSize.y, interThreshold,
                interMinAboveThreshold);
            // pairScoresCpu <-- pairScoresGpu
            hipMemcpy(pairScoresCpu.getPtr(), pairScoresGpuPtr, totalComputations * sizeof(T),
                       hipMemcpyDeviceToHost);

            // New code
            // Get pair connections and their scores
            const auto pairConnections = pafPtrIntoVector(
                pairScoresCpu, peaksPtr, maxPeaks, bodyPartPairs, numberBodyPartPairs);
            const auto peopleVector = pafVectorIntoPeopleVector(
                pairConnections, peaksPtr, maxPeaks, bodyPartPairs, numberBodyParts);

            // // Old code
            // // Get pair connections and their scores
            // // std::vector<std::pair<std::vector<int>, double>> refers to:
            // //     - std::vector<int>: [body parts locations, #body parts found]
            // //     - double: person subset score
            // const T* const tNullptr = nullptr;
            // const auto peopleVector = createPeopleVector(
            //     tNullptr, peaksPtr, poseModel, heatMapSize, maxPeaks, interThreshold, interMinAboveThreshold,
            //     bodyPartPairs, numberBodyParts, numberBodyPartPairs, pairScoresCpu);

            // Delete people below the following thresholds:
                // a) minSubsetCnt: removed if less than minSubsetCnt body parts
                // b) minSubsetScore: removed if global score smaller than this
                // c) maxPeaks (POSE_MAX_PEOPLE): keep first maxPeaks people above thresholds
            int numberPeople;
            std::vector<int> validSubsetIndexes;
            validSubsetIndexes.reserve(fastMin((size_t)maxPeaks, peopleVector.size()));
            removePeopleBelowThresholds(validSubsetIndexes, numberPeople, peopleVector, numberBodyParts, minSubsetCnt,
                                        minSubsetScore, maxPeaks, maximizePositives);

            // Fill and return poseKeypoints
            peopleVectorToPeopleArray(poseKeypoints, poseScores, scaleFactor, peopleVector, validSubsetIndexes,
                                      peaksPtr, numberPeople, numberBodyParts, numberBodyPartPairs);

            // Sanity check
            cudaCheck(__LINE__, __FUNCTION__, __FILE__);
        }
        catch (const std::exception& e)
        {
            error(e.what(), __LINE__, __FUNCTION__, __FILE__);
        }
    }

    template void connectBodyPartsGpu(
        Array<float>& poseKeypoints, Array<float>& poseScores, const float* const heatMapGpuPtr,
        const float* const peaksPtr, const PoseModel poseModel, const Point<int>& heatMapSize, const int maxPeaks,
        const float interMinAboveThreshold, const float interThreshold, const int minSubsetCnt,
        const float minSubsetScore, const float scaleFactor, const bool maximizePositives,
        Array<float> pairScoresCpu, float* pairScoresGpuPtr, const unsigned int* const bodyPartPairsGpuPtr,
        const unsigned int* const mapIdxGpuPtr, const float* const peaksGpuPtr);
    template void connectBodyPartsGpu(
        Array<double>& poseKeypoints, Array<double>& poseScores, const double* const heatMapGpuPtr,
        const double* const peaksPtr, const PoseModel poseModel, const Point<int>& heatMapSize, const int maxPeaks,
        const double interMinAboveThreshold, const double interThreshold, const int minSubsetCnt,
        const double minSubsetScore, const double scaleFactor, const bool maximizePositives,
        Array<double> pairScoresCpu, double* pairScoresGpuPtr, const unsigned int* const bodyPartPairsGpuPtr,
        const unsigned int* const mapIdxGpuPtr, const double* const peaksGpuPtr);
}
