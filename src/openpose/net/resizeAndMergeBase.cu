#include "hip/hip_runtime.h"
#include <openpose/gpu/cuda.hpp>
#include <openpose/gpu/cuda.hu>
#include <openpose/net/resizeAndMergeBase.hpp>

namespace op
{
    const auto THREADS_PER_BLOCK_1D = 16u;

    template <typename T>
    __global__ void resizeKernel(
        T* targetPtr, const T* const sourcePtr, const int sourceWidth, const int sourceHeight, const int targetWidth,
        const int targetHeight)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;

        if (x < targetWidth && y < targetHeight)
        {
            const T xSource = (x + T(0.5f)) * sourceWidth / T(targetWidth) - T(0.5f);
            const T ySource = (y + T(0.5f)) * sourceHeight / T(targetHeight) - T(0.5f);
            targetPtr[y*targetWidth+x] = bicubicInterpolate(
                sourcePtr, xSource, ySource, sourceWidth, sourceHeight, sourceWidth);
        }
    }

    template <typename T>
    __global__ void resizeAllKernel(
        T* targetPtr, const T* const sourcePtr, const int sourceWidth, const int sourceHeight, const int targetWidth,
        const int targetHeight, const int channels)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
        const auto channel = (blockIdx.z * blockDim.z) + threadIdx.z;

        const auto sourceArea = sourceWidth * sourceHeight;
        const auto targetArea = targetWidth * targetHeight;

        if (x < targetWidth && y < targetHeight && channel < channels)
        {
            const T xSource = (x + T(0.5f)) * sourceWidth / T(targetWidth) - T(0.5f);
            const T ySource = (y + T(0.5f)) * sourceHeight / T(targetHeight) - T(0.5f);
            const T* sourcePtrChannel = sourcePtr + channel * sourceArea;
            targetPtr[channel * targetArea + y*targetWidth+x] = bicubicInterpolate(
                sourcePtrChannel, xSource, ySource, sourceWidth, sourceHeight, sourceWidth);
        }
    }


        template <typename T>
    __global__ void resizeAllKernelShared(
        T* targetPtr, const T* const sourcePtr, const int sourceWidth, const int sourceHeight, const int targetWidth,
        const int targetHeight, const int channels, const unsigned int rescaleFactor)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
        const auto channel = (blockIdx.z * blockDim.z) + threadIdx.z;
        
        const auto minTargetX = blockIdx.x * rescaleFactor;
        const auto maxTargetX = ((blockIdx.x + 1) * rescaleFactor) - 1;

        const auto minTargetY = blockIdx.y * rescaleFactor;
        const auto maxTargetY = ((blockIdx.y + 1) * rescaleFactor) - 1;

        const auto minSourceX = (minTargetY + T(0.5f)) * sourceWidth / T(targetWidth) - T(0.5f);;
        const auto minSourceY = (minTargetY + T(0.5f)) * sourceHeight / T(targetHeight) - T(0.5f);

        const auto maxSourceX = (maxTargetX + T(0.5f)) * sourceWidth / T(targetWidth) - T(0.5f);
        const auto maxSourceY = (maxTargetY + T(0.5f)) * sourceHeight / T(targetHeight) - T(0.5f);
       
        //__shared__ sourcePtrsShared float[49]; 

        const auto sourceArea = sourceWidth * sourceHeight;
        const auto targetArea = targetWidth * targetHeight;

        if (threadIdx.x == 0) 
        {
            //printf("minX, minY: %f, %f | maxX, maxY: %f, %d\f", minSourceX, minSourceY, maxSourceX, maxSourceY);
            if (maxSourceX - minSourceX != 7) {
                printf("wahooo");
            }
            if (maxSourceY - minSourceY != 7) {
                printf("blaaah");
            }
            // for (auto ySource = minTargetY; ySource < maxSourceY; ySource++)
            // {
            //     for (auto xSource = minTargetX; xSource < maxSourceX; xSource++) 
            //     }
            //         sourcePtrsShared[rescaleFactor * ySource]
            //     }   
            // }
            
        }
        // wait here until shared memory has been loaded
        //__syncthreads();

        if (x < targetWidth && y < targetHeight && channel < channels) 
        {
            const T xSource = (x + T(0.5f)) * sourceWidth / T(targetWidth) - T(0.5f);
            const T ySource = (y + T(0.5f)) * sourceHeight / T(targetHeight) - T(0.5f);
            const T* sourcePtrChannel = sourcePtr + channel * sourceArea;
            targetPtr[channel * targetArea + y*targetWidth+x] = bicubicInterpolate(
                sourcePtrChannel, xSource, ySource, sourceWidth, sourceHeight, sourceWidth);
        }
    }

    template <typename T>
    __global__ void resizeKernelAndAdd(T* targetPtr, const T* const sourcePtr, const T scaleWidth,
                                       const T scaleHeight, const int sourceWidth, const int sourceHeight,
                                       const int targetWidth, const int targetHeight)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;

        if (x < targetWidth && y < targetHeight)
        {
            const T xSource = (x + T(0.5f)) / scaleWidth - T(0.5f);
            const T ySource = (y + T(0.5f)) / scaleHeight - T(0.5f);
            targetPtr[y*targetWidth+x] += bicubicInterpolate(sourcePtr, xSource, ySource, sourceWidth, sourceHeight,
                                                             sourceWidth);
        }
    }

    template <typename T>
    __global__ void resizeKernelAndAverage(T* targetPtr, const T* const sourcePtr, const T scaleWidth,
                                           const T scaleHeight, const int sourceWidth, const int sourceHeight,
                                           const int targetWidth, const int targetHeight, const int counter)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;

        if (x < targetWidth && y < targetHeight)
        {
            const T xSource = (x + T(0.5f)) / scaleWidth - T(0.5f);
            const T ySource = (y + T(0.5f)) / scaleHeight - T(0.5f);
            const auto interpolated = bicubicInterpolate(sourcePtr, xSource, ySource, sourceWidth, sourceHeight,
                                                         sourceWidth);
            auto& targetPixel = targetPtr[y*targetWidth+x];
            targetPixel = (targetPixel + interpolated) / T(counter);
        }
    }

    template <typename T>
    void resizeAndMergeGpu(T* targetPtr, const std::vector<const T*>& sourcePtrs, const std::array<int, 4>& targetSize,
                           const std::vector<std::array<int, 4>>& sourceSizes,
                           const std::vector<T>& scaleInputToNetInputs)
    {
        try
        {
            // Sanity checks
            if (sourceSizes.empty())
                error("sourceSizes cannot be empty.", __LINE__, __FUNCTION__, __FILE__);
            if (sourcePtrs.size() != sourceSizes.size() || sourceSizes.size() != scaleInputToNetInputs.size())
                error("Size(sourcePtrs) must match size(sourceSizes) and size(scaleInputToNetInputs). Currently: "
                      + std::to_string(sourcePtrs.size()) + " vs. " + std::to_string(sourceSizes.size()) + " vs. "
                      + std::to_string(scaleInputToNetInputs.size()) + ".", __LINE__, __FUNCTION__, __FILE__);

            // Parameters
            const auto channels = targetSize[1];
            const auto targetHeight = targetSize[2];
            const auto targetWidth = targetSize[3];
            const dim3 threadsPerBlock{THREADS_PER_BLOCK_1D, THREADS_PER_BLOCK_1D};
            const dim3 numBlocks{getNumberCudaBlocks(targetWidth, threadsPerBlock.x),
                                 getNumberCudaBlocks(targetHeight, threadsPerBlock.y)};
            const auto& sourceSize = sourceSizes[0];
            const auto sourceHeight = sourceSize[2];
            const auto sourceWidth = sourceSize[3];

            // No multi-scale merging or no merging required
            if (sourceSizes.size() == 1)
            {
                const auto num = sourceSize[0];
                if (targetSize[0] > 1 || num == 1)
                {
const auto REPS = 1;
// const auto REPS = 1;
double timeNormalize0 = 0.;
double timeNormalize1 = 0.;
double timeNormalize2 = 0.;
double timeNormalize3 = 0.;
double timeNormalize4 = 0.;
double timeNormalize5 = 0.;
// OP_CUDA_PROFILE_INIT(5);
//                     // Option a)
//                     const auto sourceChannelOffset = sourceHeight * sourceWidth;
//                     const auto targetChannelOffset = targetWidth * targetHeight;
//                     for (auto n = 0; n < num; n++)
//                     {
//                         const auto offsetBase = n*channels;
//                         for (auto c = 0 ; c < channels ; c++)
//                         {
//                             const auto offset = offsetBase + c;
//                             resizeKernel<<<numBlocks, threadsPerBlock>>>(targetPtr + offset * targetChannelOffset,
//                                                                          sourcePtrs.at(0) + offset * sourceChannelOffset,
//                                                                          sourceWidth, sourceHeight, targetWidth,
//                                                                          targetHeight);
//                         }
//                     }
// OP_CUDA_PROFILE_END(timeNormalize0, 1e3, 5);
// OP_CUDA_PROFILE_INIT(REPS);
//                     // Option a)
//                     const auto sourceChannelOffset = sourceHeight * sourceWidth;
//                     const auto targetChannelOffset = targetWidth * targetHeight;
//                     for (auto n = 0; n < num; n++)
//                     {
//                         const auto offsetBase = n*channels;
//                         for (auto c = 0 ; c < channels ; c++)
//                         {
//                             const auto offset = offsetBase + c;
//                             resizeKernel<<<numBlocks, threadsPerBlock>>>(targetPtr + offset * targetChannelOffset,
//                                                                          sourcePtrs.at(0) + offset * sourceChannelOffset,
//                                                                          sourceWidth, sourceHeight, targetWidth,
//                                                                          targetHeight);
//                         }
//                     }
// OP_CUDA_PROFILE_END(timeNormalize1, 1e3, REPS);
// OP_CUDA_PROFILE_INIT(REPS);
//                     // Option a)
//                     const dim3 threadsPerBlock{512, 1};
//                     const dim3 numBlocks{getNumberCudaBlocks(targetWidth, threadsPerBlock.x),
//                                          getNumberCudaBlocks(targetHeight, threadsPerBlock.y)};
//                     const auto sourceChannelOffset = sourceHeight * sourceWidth;
//                     const auto targetChannelOffset = targetWidth * targetHeight;
//                     for (auto n = 0; n < num; n++)
//                     {
//                         const auto offsetBase = n*channels;
//                         for (auto c = 0 ; c < channels ; c++)
//                         {
//                             const auto offset = offsetBase + c;
//                             resizeKernel<<<numBlocks, threadsPerBlock>>>(targetPtr + offset * targetChannelOffset,
//                                                                          sourcePtrs.at(0) + offset * sourceChannelOffset,
//                                                                          sourceWidth, sourceHeight, targetWidth,
//                                                                          targetHeight);
//                         }
//                     }
// OP_CUDA_PROFILE_END(timeNormalize2, 1e3, REPS);
// OP_CUDA_PROFILE_INIT(REPS);
//                     // Option b)
//                     const dim3 threadsPerBlock{512, 1, 1};
//                     const dim3 numBlocks{getNumberCudaBlocks(targetWidth, threadsPerBlock.x),
//                                          getNumberCudaBlocks(targetHeight, threadsPerBlock.y),
//                                          getNumberCudaBlocks(num * channels, threadsPerBlock.z)};
//                     resizeAllKernel<<<numBlocks, threadsPerBlock>>>(
//                         targetPtr, sourcePtrs.at(0), sourceWidth, sourceHeight, targetWidth, targetHeight,
//                         num * channels);
// OP_CUDA_PROFILE_END(timeNormalize3, 1e3, REPS);
// OP_CUDA_PROFILE_INIT(REPS);
//                     // Option b)
//                     const dim3 threadsPerBlock{THREADS_PER_BLOCK_1D, THREADS_PER_BLOCK_1D, 1};
//                     const dim3 numBlocks{getNumberCudaBlocks(targetWidth, threadsPerBlock.x),
//                                          getNumberCudaBlocks(targetHeight, threadsPerBlock.y),
//                                          getNumberCudaBlocks(num * channels, threadsPerBlock.z)};
//                     resizeAllKernel<<<numBlocks, threadsPerBlock>>>(
//                         targetPtr, sourcePtrs.at(0), sourceWidth, sourceHeight, targetWidth, targetHeight,
//                         num * channels);
// OP_CUDA_PROFILE_END(timeNormalize4, 1e3, REPS);
OP_CUDA_PROFILE_INIT(REPS);
                    // Option b)
                    const auto rescaleFactor = (unsigned int) std::ceil((float)(targetHeight) / (float)(sourceHeight));

                    const dim3 threadsPerBlock{rescaleFactor, rescaleFactor, 1};
                    const dim3 numBlocks{getNumberCudaBlocks(targetWidth, threadsPerBlock.x),
                                         getNumberCudaBlocks(targetHeight, threadsPerBlock.y),
                                         getNumberCudaBlocks(num * channels, threadsPerBlock.z)};
                    resizeAllKernelShared<<<numBlocks, threadsPerBlock>>>(
                        targetPtr, sourcePtrs.at(0), sourceWidth, sourceHeight, targetWidth, targetHeight,
                        num * channels, rescaleFactor);
OP_CUDA_PROFILE_END(timeNormalize5, 1e3, REPS);
log("  Res1(ori)=" + std::to_string(timeNormalize1) + "ms");
log("  Res2(ori)=" + std::to_string(timeNormalize2) + "ms");
log("  Res3(new)=" + std::to_string(timeNormalize3) + "ms");
log("  Res4(new)=" + std::to_string(timeNormalize4) + "ms");
log("  Res5(new)=" + std::to_string(timeNormalize5) + "ms");
                }
                // Old inefficient multi-scale merging
                else
                    error("It should never reache this point. Notify us otherwise.", __LINE__, __FUNCTION__, __FILE__);
            }
            // Multi-scaling merging
            else
            {
                const auto targetChannelOffset = targetWidth * targetHeight;
                hipMemset(targetPtr, 0, channels*targetChannelOffset * sizeof(T));
                const auto scaleToMainScaleWidth = targetWidth / T(sourceWidth);
                const auto scaleToMainScaleHeight = targetHeight / T(sourceHeight);

                for (auto i = 0u ; i < sourceSizes.size(); i++)
                {
                    const auto& currentSize = sourceSizes.at(i);
                    const auto currentHeight = currentSize[2];
                    const auto currentWidth = currentSize[3];
                    const auto sourceChannelOffset = currentHeight * currentWidth;
                    const auto scaleInputToNet = scaleInputToNetInputs[i] / scaleInputToNetInputs[0];
                    const auto scaleWidth = scaleToMainScaleWidth / scaleInputToNet;
                    const auto scaleHeight = scaleToMainScaleHeight / scaleInputToNet;
                    // All but last image --> add
                    if (i < sourceSizes.size() - 1)
                    {
                        for (auto c = 0 ; c < channels ; c++)
                        {
                            resizeKernelAndAdd<<<numBlocks, threadsPerBlock>>>(
                                targetPtr + c * targetChannelOffset, sourcePtrs[i] + c * sourceChannelOffset,
                                scaleWidth, scaleHeight, currentWidth, currentHeight, targetWidth,
                                targetHeight
                            );
                        }
                    }
                    // Last image --> average all
                    else
                    {
                        for (auto c = 0 ; c < channels ; c++)
                        {
                            resizeKernelAndAverage<<<numBlocks, threadsPerBlock>>>(
                                targetPtr + c * targetChannelOffset, sourcePtrs[i] + c * sourceChannelOffset,
                                scaleWidth, scaleHeight, currentWidth, currentHeight, targetWidth,
                                targetHeight, (int)sourceSizes.size()
                            );
                        }
                    }
                }
            }

            cudaCheck(__LINE__, __FUNCTION__, __FILE__);
        }
        catch (const std::exception& e)
        {
            error(e.what(), __LINE__, __FUNCTION__, __FILE__);
        }
    }

    template void resizeAndMergeGpu(
        float* targetPtr, const std::vector<const float*>& sourcePtrs, const std::array<int, 4>& targetSize,
        const std::vector<std::array<int, 4>>& sourceSizes, const std::vector<float>& scaleInputToNetInputs);
    template void resizeAndMergeGpu(
        double* targetPtr, const std::vector<const double*>& sourcePtrs, const std::array<int, 4>& targetSize,
        const std::vector<std::array<int, 4>>& sourceSizes, const std::vector<double>& scaleInputToNetInputs);
}
