#include "hip/hip_runtime.h"
#include <openpose/gpu/cuda.hpp>
#include <openpose/gpu/cuda.hu>
#include <openpose/net/resizeAndMergeBase.hpp>

namespace op
{
    const auto THREADS_PER_BLOCK_1D = 16u;

    // template <typename T>
    // __global__ void resizeKernelOld(
    //     T* targetPtr, const T* const sourcePtr, const int sourceWidth, const int sourceHeight, const int targetWidth,
    //     const int targetHeight)
    // {
    //     const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
    //     const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;

    //     if (x < targetWidth && y < targetHeight)
    //     {
    //         const T xSource = (x + T(0.5f)) * sourceWidth / T(targetWidth) - T(0.5f);
    //         const T ySource = (y + T(0.5f)) * sourceHeight / T(targetHeight) - T(0.5f);
    //         targetPtr[y*targetWidth+x] = bicubicInterpolate(
    //             sourcePtr, xSource, ySource, sourceWidth, sourceHeight, sourceWidth);
    //     }
    // }

    template <typename T>
    __global__ void resizeKernel(
        T* targetPtr, const T* const sourcePtr, const int sourceWidth, const int sourceHeight, const int targetWidth,
        const int targetHeight, const int channels)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
        const auto channel = (blockIdx.z * blockDim.z) + threadIdx.z;

        const auto sourceArea = sourceWidth * sourceHeight;
        const auto targetArea = targetWidth * targetHeight;

        if (x < targetWidth && y < targetHeight && channel < channels)
        {
            const T xSource = (x + T(0.5f)) * sourceWidth / T(targetWidth) - T(0.5f);
            const T ySource = (y + T(0.5f)) * sourceHeight / T(targetHeight) - T(0.5f);
            const T* sourcePtrChannel = sourcePtr + channel * sourceArea;
            targetPtr[channel * targetArea + y*targetWidth+x] = bicubicInterpolate(
                sourcePtrChannel, xSource, ySource, sourceWidth, sourceHeight, sourceWidth);
        }
    }

    template <typename T>
    __global__ void resizeAndPadKernel(
        T* targetPtr, const T* const sourcePtr, const int sourceWidth, const int sourceHeight, const int targetWidth,
        const int targetHeight, const float rescaleFactor, const int channels)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
        const auto channel = (blockIdx.z * blockDim.z) + threadIdx.z;

        const auto sourceArea = sourceWidth * sourceHeight;
        const auto targetArea = targetWidth * targetHeight;

        if (x < targetWidth && y < targetHeight && channel < channels)
        {
            const T xSource = (x + T(0.5f)) * 1.0 / T(rescaleFactor) - T(0.5f);
            const T ySource = (y + T(0.5f)) * 1.0 / T(rescaleFactor) - T(0.5f);
            const T* sourcePtrChannel = sourcePtr + channel * sourceArea;
            if (x < sourceWidth * rescaleFactor && y < sourceHeight * rescaleFactor)
                targetPtr[channel * targetArea + y*targetWidth+x] = bicubicInterpolate(
                    sourcePtrChannel, xSource, ySource, sourceWidth, sourceHeight, sourceWidth);
            else
                targetPtr[channel * targetArea + y*targetWidth+x] = 0;
        }
    }


        template <typename T>
    __global__ void resize8TimesKernel(
        T* targetPtr, const T* const sourcePtr, const int sourceWidth, const int sourceHeight, const int targetWidth,
        const int targetHeight, const unsigned int rescaleFactor)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
        const auto channel = (blockIdx.z * blockDim.z) + threadIdx.z;

        // Load shared memory
        // If resize >= 5, then #threads per block >= # elements of shared memory
        __shared__ T sourcePtrShared[25];
        const auto sharedLoadId = threadIdx.x + rescaleFactor*threadIdx.y;
        if (sharedLoadId < 25)
        {
            const auto minTargetX = blockIdx.x * rescaleFactor;
            const auto minSourceX = (minTargetX + T(0.5f)) * sourceWidth / T(targetWidth) - T(0.5f);
            const auto minSourceXInt_1 = int(minSourceX+ 1e-5);
            const auto minSourceXInt = minSourceXInt_1 - 1;
            const auto minTargetY = blockIdx.y * rescaleFactor;
            const auto minSourceY = (minTargetY + T(0.5f)) * sourceHeight / T(targetHeight) - T(0.5f);
            const auto minSourceYInt_1 = int(minSourceY  + 1e-5);
            const auto minSourceYInt = minSourceYInt_1 - 1;

            const auto yClean = fastTruncateCuda(int(minSourceYInt+sharedLoadId/5 + 1e-5), 0, sourceHeight - 1);
            const auto xClean = fastTruncateCuda(int(minSourceXInt+sharedLoadId%5 + 1e-5), 0, sourceWidth - 1);
            const auto sourceArea = sourceWidth * sourceHeight;
            const T* sourcePtrChannel = sourcePtr + channel * sourceArea;
            sourcePtrShared[sharedLoadId] = sourcePtrChannel[yClean * sourceWidth + xClean];
        }
        // Wait here until shared memory has been loaded
        __syncthreads();

        if (x < targetWidth && y < targetHeight)
        {
            const auto targetArea = targetWidth * targetHeight;
            const T xSource = (x + T(0.5f)) * sourceWidth / T(targetWidth) - T(0.5f);
            const T ySource = (y + T(0.5f)) * sourceHeight / T(targetHeight) - T(0.5f);
            targetPtr[channel * targetArea + y*targetWidth+x] = bicubicInterpolate8Times(
                sourcePtrShared, xSource, ySource, sourceWidth, sourceHeight, sourceWidth, threadIdx.x, threadIdx.y);
        }
    }

    template <typename T>
    __global__ void resizeKernelAndAdd(T* targetPtr, const T* const sourcePtr, const T scaleWidth,
                                       const T scaleHeight, const int sourceWidth, const int sourceHeight,
                                       const int targetWidth, const int targetHeight)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;

        if (x < targetWidth && y < targetHeight)
        {
            const T xSource = (x + T(0.5f)) / scaleWidth - T(0.5f);
            const T ySource = (y + T(0.5f)) / scaleHeight - T(0.5f);
            targetPtr[y*targetWidth+x] += bicubicInterpolate(sourcePtr, xSource, ySource, sourceWidth, sourceHeight,
                                                             sourceWidth);
        }
    }

    template <typename T>
    __global__ void resizeKernelAndAverage(T* targetPtr, const T* const sourcePtr, const T scaleWidth,
                                           const T scaleHeight, const int sourceWidth, const int sourceHeight,
                                           const int targetWidth, const int targetHeight, const int counter)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;

        if (x < targetWidth && y < targetHeight)
        {
            const T xSource = (x + T(0.5f)) / scaleWidth - T(0.5f);
            const T ySource = (y + T(0.5f)) / scaleHeight - T(0.5f);
            const auto interpolated = bicubicInterpolate(sourcePtr, xSource, ySource, sourceWidth, sourceHeight,
                                                         sourceWidth);
            auto& targetPixel = targetPtr[y*targetWidth+x];
            targetPixel = (targetPixel + interpolated) / T(counter);
        }
    }

    __global__ void reorderAndCastKernel(
        float* targetPtr, const unsigned char* const srcPtr, const int width, const int height)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
        const auto c = (blockIdx.z * blockDim.z) + threadIdx.z;
        if (x < width && y < height)
        {
            const auto channels = 3;
            const auto originFramePtrOffsetY = y * width;
            const auto channelOffset = c * width * height;
            const auto targetIndex = channelOffset + y * width + x;
            const auto srcIndex = (originFramePtrOffsetY + x) * channels + c;
            targetPtr[targetIndex] =  float(srcPtr[srcIndex]) * (1/256.f) - 0.5f;
        }
    }

    void reorderAndCast(float* targetPtr, const unsigned char* const srcPtr, const int width, const int height)
    {
        const dim3 threadsPerBlock{32, 1, 1};
        const dim3 numBlocks{
            getNumberCudaBlocks(width, threadsPerBlock.x),
            getNumberCudaBlocks(height, threadsPerBlock.y),
            getNumberCudaBlocks(3, threadsPerBlock.z)};
        reorderAndCastKernel<<<numBlocks, threadsPerBlock>>>(targetPtr, srcPtr, width, height);
    }

    void resizeAndMergeRGBGPU(
        float* targetPtr, const float* const srcPtr, const int sourceWidth, const int sourceHeight,
        const int targetWidth, const int targetHeight, const float scaleFactor)

    {
        const auto channels = 3;
        const dim3 threadsPerBlock{THREADS_PER_BLOCK_1D, THREADS_PER_BLOCK_1D, 1};
        const dim3 numBlocks{
            getNumberCudaBlocks(targetWidth, threadsPerBlock.x),
            getNumberCudaBlocks(targetHeight, threadsPerBlock.y),
            getNumberCudaBlocks(channels, threadsPerBlock.z)};

        resizeAndPadKernel<<<numBlocks, threadsPerBlock>>>(
            targetPtr, srcPtr, sourceWidth, sourceHeight, targetWidth, targetHeight, scaleFactor, channels);
    }

    template <typename T>
    void resizeAndMergeGpu(
        T* targetPtr, const std::vector<const T*>& sourcePtrs, const std::array<int, 4>& targetSize,
        const std::vector<std::array<int, 4>>& sourceSizes, const std::vector<T>& scaleInputToNetInputs)
    {
        try
        {
            // Sanity checks
            if (sourceSizes.empty())
                error("sourceSizes cannot be empty.", __LINE__, __FUNCTION__, __FILE__);
            if (sourcePtrs.size() != sourceSizes.size() || sourceSizes.size() != scaleInputToNetInputs.size())
                error("Size(sourcePtrs) must match size(sourceSizes) and size(scaleInputToNetInputs). Currently: "
                      + std::to_string(sourcePtrs.size()) + " vs. " + std::to_string(sourceSizes.size()) + " vs. "
                      + std::to_string(scaleInputToNetInputs.size()) + ".", __LINE__, __FUNCTION__, __FILE__);

            // Parameters
            const auto channels = targetSize[1];
            const auto targetHeight = targetSize[2];
            const auto targetWidth = targetSize[3];
            const dim3 threadsPerBlock{THREADS_PER_BLOCK_1D, THREADS_PER_BLOCK_1D};
            const dim3 numBlocks{getNumberCudaBlocks(targetWidth, threadsPerBlock.x),
                                 getNumberCudaBlocks(targetHeight, threadsPerBlock.y)};
            const auto& sourceSize = sourceSizes[0];
            const auto sourceHeight = sourceSize[2];
            const auto sourceWidth = sourceSize[3];

            // No multi-scale merging or no merging required
            if (sourceSizes.size() == 1)
            {
                const auto num = sourceSize[0];
                if (targetSize[0] > 1 || num == 1)
                {
                    // // Profiling code
                    // const auto REPS = 250;
                    // double timeNormalize0 = 0.;
                    // double timeNormalize1 = 0.;
                    // double timeNormalize2 = 0.;
                    // double timeNormalize3 = 0.;
                    // // Non-optimized function
                    // OP_CUDA_PROFILE_INIT(REPS);
                    // const auto sourceChannelOffset = sourceHeight * sourceWidth;
                    // const auto targetChannelOffset = targetWidth * targetHeight;
                    // for (auto n = 0; n < num; n++)
                    // {
                    //     const auto offsetBase = n*channels;
                    //     for (auto c = 0 ; c < channels ; c++)
                    //     {
                    //         const auto offset = offsetBase + c;
                    //         resizeKernelOld<<<numBlocks, threadsPerBlock>>>(
                    //             targetPtr + offset * targetChannelOffset,
                    //             sourcePtrs.at(0) + offset * sourceChannelOffset,
                    //             sourceWidth, sourceHeight, targetWidth, targetHeight);
                    //     }
                    // }
                    // OP_CUDA_PROFILE_END(timeNormalize1, 1e3, REPS);

                    // Optimized function for any resize size (suboptimal for 8x resize)
                    // OP_CUDA_PROFILE_INIT(REPS);
                    const dim3 threadsPerBlock{THREADS_PER_BLOCK_1D, THREADS_PER_BLOCK_1D, 1};
                    const dim3 numBlocks{getNumberCudaBlocks(targetWidth, threadsPerBlock.x),
                                         getNumberCudaBlocks(targetHeight, threadsPerBlock.y),
                                         getNumberCudaBlocks(num * channels, threadsPerBlock.z)};
                    resizeKernel<<<numBlocks, threadsPerBlock>>>(
                        targetPtr, sourcePtrs.at(0), sourceWidth, sourceHeight, targetWidth, targetHeight,
                        num * channels);
                    // OP_CUDA_PROFILE_END(timeNormalize2, 1e3, REPS);

                    // // Optimized function for 8x resize
                    // OP_CUDA_PROFILE_INIT(REPS);
                    // if (targetWidth / sourceWidth != 8 || targetHeight / sourceHeight != 8)
                    //     error("Kernel only implemented for 8x resize. Notify us if this error appears.",
                    //         __LINE__, __FUNCTION__, __FILE__);
                    // const auto rescaleFactor = (unsigned int) std::ceil((float)(targetHeight) / (float)(sourceHeight));

                    // const dim3 threadsPerBlock{rescaleFactor, rescaleFactor, 1};
                    // const dim3 numBlocks{getNumberCudaBlocks(targetWidth, threadsPerBlock.x),
                    //                      getNumberCudaBlocks(targetHeight, threadsPerBlock.y),
                    //                      getNumberCudaBlocks(num * channels, threadsPerBlock.z)};
                    // resize8TimesKernel<<<numBlocks, threadsPerBlock>>>(
                    //     targetPtr, sourcePtrs.at(0), sourceWidth, sourceHeight, targetWidth, targetHeight, rescaleFactor);
                    // OP_CUDA_PROFILE_END(timeNormalize3, 1e3, REPS);

                    // Profiling code
                    // log("  Res(ori)=" + std::to_string(timeNormalize1) + "ms");
                    // log("  Res(new)=" + std::to_string(timeNormalize2) + "ms");
                    // log("  Res(new8x)=" + std::to_string(timeNormalize3) + "ms");
                }
                // Old inefficient multi-scale merging
                else
                    error("It should never reache this point. Notify us otherwise.", __LINE__, __FUNCTION__, __FILE__);
            }
            // Multi-scaling merging
            else
            {
                const auto targetChannelOffset = targetWidth * targetHeight;
                hipMemset(targetPtr, 0, channels*targetChannelOffset * sizeof(T));
                const auto scaleToMainScaleWidth = targetWidth / T(sourceWidth);
                const auto scaleToMainScaleHeight = targetHeight / T(sourceHeight);

                for (auto i = 0u ; i < sourceSizes.size(); i++)
                {
                    const auto& currentSize = sourceSizes.at(i);
                    const auto currentHeight = currentSize[2];
                    const auto currentWidth = currentSize[3];
                    const auto sourceChannelOffset = currentHeight * currentWidth;
                    const auto scaleInputToNet = scaleInputToNetInputs[i] / scaleInputToNetInputs[0];
                    const auto scaleWidth = scaleToMainScaleWidth / scaleInputToNet;
                    const auto scaleHeight = scaleToMainScaleHeight / scaleInputToNet;
                    // All but last image --> add
                    if (i < sourceSizes.size() - 1)
                    {
                        for (auto c = 0 ; c < channels ; c++)
                        {
                            resizeKernelAndAdd<<<numBlocks, threadsPerBlock>>>(
                                targetPtr + c * targetChannelOffset, sourcePtrs[i] + c * sourceChannelOffset,
                                scaleWidth, scaleHeight, currentWidth, currentHeight, targetWidth,
                                targetHeight
                            );
                        }
                    }
                    // Last image --> average all
                    else
                    {
                        for (auto c = 0 ; c < channels ; c++)
                        {
                            resizeKernelAndAverage<<<numBlocks, threadsPerBlock>>>(
                                targetPtr + c * targetChannelOffset, sourcePtrs[i] + c * sourceChannelOffset,
                                scaleWidth, scaleHeight, currentWidth, currentHeight, targetWidth,
                                targetHeight, (int)sourceSizes.size()
                            );
                        }
                    }
                }
            }

            cudaCheck(__LINE__, __FUNCTION__, __FILE__);
        }
        catch (const std::exception& e)
        {
            error(e.what(), __LINE__, __FUNCTION__, __FILE__);
        }
    }

    template void resizeAndMergeGpu(
        float* targetPtr, const std::vector<const float*>& sourcePtrs, const std::array<int, 4>& targetSize,
        const std::vector<std::array<int, 4>>& sourceSizes, const std::vector<float>& scaleInputToNetInputs);
    template void resizeAndMergeGpu(
        double* targetPtr, const std::vector<const double*>& sourcePtrs, const std::array<int, 4>& targetSize,
        const std::vector<std::array<int, 4>>& sourceSizes, const std::vector<double>& scaleInputToNetInputs);
}
