#include "hip/hip_runtime.h"
#include <openpose/gpu/cuda.hpp>
#include <openpose/gpu/cuda.hu>
#include <openpose/net/resizeAndMergeBase.hpp>

namespace op
{
    const auto THREADS_PER_BLOCK_1D = 16u;

    template <typename T>
    __global__ void resizeKernel(
        T* targetPtr, const T* const sourcePtr, const int sourceWidth, const int sourceHeight, const int targetWidth,
        const int targetHeight)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;

        if (x < targetWidth && y < targetHeight)
        {
            const T xSource = (x + T(0.5f)) * sourceWidth / T(targetWidth) - T(0.5f);
            const T ySource = (y + T(0.5f)) * sourceHeight / T(targetHeight) - T(0.5f);
            targetPtr[y*targetWidth+x] = bicubicInterpolate(
                sourcePtr, xSource, ySource, sourceWidth, sourceHeight, sourceWidth);
        }
    }

    template <typename T>
    __global__ void resizeAllKernel(
        T* targetPtr, const T* const sourcePtr, const int sourceWidth, const int sourceHeight, const int targetWidth,
        const int targetHeight, const int channels)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
        const auto channel = (blockIdx.z * blockDim.z) + threadIdx.z;

        const auto sourceArea = sourceWidth * sourceHeight;
        const auto targetArea = targetWidth * targetHeight;

        if (x < targetWidth && y < targetHeight && channel < channels)
        {
            const T xSource = (x + T(0.5f)) * sourceWidth / T(targetWidth) - T(0.5f);
            const T ySource = (y + T(0.5f)) * sourceHeight / T(targetHeight) - T(0.5f);
            const T* sourcePtrChannel = sourcePtr + channel * sourceArea;
            targetPtr[channel * targetArea + y*targetWidth+x] = bicubicInterpolate(
                sourcePtrChannel, xSource, ySource, sourceWidth, sourceHeight, sourceWidth);
        }
    }


        template <typename T>
    __global__ void resizeAllKernelShared(
        T* targetPtr, const T* const sourcePtr, const int sourceWidth, const int sourceHeight, const int targetWidth,
        const int targetHeight, const int channels, const unsigned int rescaleFactor)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;
        const auto channel = (blockIdx.z * blockDim.z) + threadIdx.z;
        
        const auto minTargetX = blockIdx.x * rescaleFactor;
        const auto maxTargetX = ((blockIdx.x + 1) * rescaleFactor) - 1;

        const auto minTargetY = blockIdx.y * rescaleFactor;
        const auto maxTargetY = ((blockIdx.y + 1) * rescaleFactor) - 1;
       
        auto minSourceX = (minTargetX + T(0.5f)) * sourceWidth / T(targetWidth) - T(0.5f);
        
        auto minSourceXInt_1 = int(minSourceX+ 1e-5);
        auto minSourceXInt = minSourceXInt_1 - 1;

        auto minSourceY = (minTargetY + T(0.5f)) * sourceHeight / T(targetHeight) - T(0.5f);
        
        auto minSourceYInt_1 = int(minSourceY  + 1e-5);
        // auto minSourceYInt = fastMaxCuda(0, minSourceYInt_1 - 1);
        auto minSourceYInt = minSourceYInt_1 - 1;
        //xIntArray[2] = fastMinCuda(sourceWidth - 1, xIntArray[1] + 1);

        auto maxSourceX = (maxTargetX + T(0.5f)) * sourceWidth / T(targetWidth) - T(0.5f);
        // auto maxSourceXInt_1 = fastTruncateCuda(int(maxSourceX + 1e-5), 0, sourceWidth - 1);
        auto maxSourceXInt_1 = int(maxSourceX + 1e-5);
        // auto maxSourceXInt = fastMinCuda(sourceWidth - 1, maxSourceXInt_1 + 2);
        auto maxSourceXInt = maxSourceXInt_1 + 2;

        auto maxSourceY = (maxTargetY + T(0.5f)) * sourceHeight / T(targetHeight) - T(0.5f);
        // auto maxSourceYInt_1 = fastTruncateCuda(int(maxSourceY + 1e-5), 0, sourceHeight - 1);
        auto maxSourceYInt_1 = int(maxSourceY + 1e-5);
        // auto maxSourceYInt = fastMinCuda(sourceHeight - 1, maxSourceYInt_1 + 2);
        auto maxSourceYInt = maxSourceYInt_1 + 2;

        __shared__ T sourcePtrShared[25]; 

        const auto sourceArea = sourceWidth * sourceHeight;
        const auto targetArea = targetWidth * targetHeight;

        // if (x == 16 && (y == 8) && channel == 0)
        // {
            
        //     printf("minTarX, minTarY: %d, %d | maxTarX, maxTarY: %d, %d\n", minTargetX, minTargetY, maxTargetX, maxTargetY);
        //     printf("minSourceXInt_1, minSourceY_Int1: %d, %d\n", minSourceXInt_1, minSourceYInt_1);
        //     printf("minX, minY: %d, %d | maxX, maxY: %d, %d\n", minSourceXInt, minSourceYInt, maxSourceXInt, maxSourceYInt);
        //     // if (maxSourceX - minSourceX != 7) {
        //     //     printf("wahooo");
        //     // }
        //     // if (maxSourceY - minSourceY != 7) {
        //     //     printf("blaaah");
        //     // }
        // }
        const T* sourcePtrChannel = sourcePtr + channel * sourceArea;
        // if resize >= 5, then #threads per block >= # elements of shared memory
        const auto sharedLoadId = threadIdx.x + rescaleFactor*threadIdx.y;
        if (sharedLoadId < 25) 
        {   
            const auto yClean = fastTruncateCuda(int(minSourceYInt+sharedLoadId/5 + 1e-5), 0, sourceHeight - 1);
            const auto xClean = fastTruncateCuda(int(minSourceXInt+sharedLoadId%5 + 1e-5), 0, sourceWidth - 1);
            sourcePtrShared[sharedLoadId] = sourcePtrChannel[yClean * sourceWidth + xClean];
        }
        // if (threadIdx.x == 0) 
        // {   
        //     auto index = 0;
        //     for (auto ySource = minSourceYInt; ySource <= maxSourceYInt; ySource++)
        //     {
        //         const auto yClean = fastTruncateCuda(int(ySource + 1e-5), 0, sourceHeight - 1);
        //         for (auto xSource = minSourceXInt; xSource <= maxSourceXInt; xSource++) 
        //         {
        //             const auto xClean = fastTruncateCuda(int(xSource + 1e-5), 0, sourceWidth - 1);
        //             //const T* sourcePtrChannel = sourcePtr + channel * sourceArea;
        //             sourcePtrShared[index] = sourcePtrChannel[yClean * sourceWidth + xClean];
        //             index ++;
        //         }   
        //     }
        // }
        // wait here until shared memory has been loaded
        __syncthreads();

        if (x < targetWidth && y < targetHeight) 
        {
            const T xSource = (x + T(0.5f)) * sourceWidth / T(targetWidth) - T(0.5f);
            const T ySource = (y + T(0.5f)) * sourceHeight / T(targetHeight) - T(0.5f);
            targetPtr[channel * targetArea + y*targetWidth+x] = bicubicInterpolateShared(
                sourcePtrShared, xSource, ySource, sourceWidth, sourceHeight, sourceWidth, threadIdx.x, threadIdx.y);
        }
    }

    template <typename T>
    __global__ void resizeKernelAndAdd(T* targetPtr, const T* const sourcePtr, const T scaleWidth,
                                       const T scaleHeight, const int sourceWidth, const int sourceHeight,
                                       const int targetWidth, const int targetHeight)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;

        if (x < targetWidth && y < targetHeight)
        {
            const T xSource = (x + T(0.5f)) / scaleWidth - T(0.5f);
            const T ySource = (y + T(0.5f)) / scaleHeight - T(0.5f);
            targetPtr[y*targetWidth+x] += bicubicInterpolate(sourcePtr, xSource, ySource, sourceWidth, sourceHeight,
                                                             sourceWidth);
        }
    }

    template <typename T>
    __global__ void resizeKernelAndAverage(T* targetPtr, const T* const sourcePtr, const T scaleWidth,
                                           const T scaleHeight, const int sourceWidth, const int sourceHeight,
                                           const int targetWidth, const int targetHeight, const int counter)
    {
        const auto x = (blockIdx.x * blockDim.x) + threadIdx.x;
        const auto y = (blockIdx.y * blockDim.y) + threadIdx.y;

        if (x < targetWidth && y < targetHeight)
        {
            const T xSource = (x + T(0.5f)) / scaleWidth - T(0.5f);
            const T ySource = (y + T(0.5f)) / scaleHeight - T(0.5f);
            const auto interpolated = bicubicInterpolate(sourcePtr, xSource, ySource, sourceWidth, sourceHeight,
                                                         sourceWidth);
            auto& targetPixel = targetPtr[y*targetWidth+x];
            targetPixel = (targetPixel + interpolated) / T(counter);
        }
    }

    template <typename T>
    void resizeAndMergeGpu(T* targetPtr, const std::vector<const T*>& sourcePtrs, const std::array<int, 4>& targetSize,
                           const std::vector<std::array<int, 4>>& sourceSizes,
                           const std::vector<T>& scaleInputToNetInputs)
    {
        try
        {
            // Sanity checks
            if (sourceSizes.empty())
                error("sourceSizes cannot be empty.", __LINE__, __FUNCTION__, __FILE__);
            if (sourcePtrs.size() != sourceSizes.size() || sourceSizes.size() != scaleInputToNetInputs.size())
                error("Size(sourcePtrs) must match size(sourceSizes) and size(scaleInputToNetInputs). Currently: "
                      + std::to_string(sourcePtrs.size()) + " vs. " + std::to_string(sourceSizes.size()) + " vs. "
                      + std::to_string(scaleInputToNetInputs.size()) + ".", __LINE__, __FUNCTION__, __FILE__);

            // Parameters
            const auto channels = targetSize[1];
            const auto targetHeight = targetSize[2];
            const auto targetWidth = targetSize[3];
            const dim3 threadsPerBlock{THREADS_PER_BLOCK_1D, THREADS_PER_BLOCK_1D};
            const dim3 numBlocks{getNumberCudaBlocks(targetWidth, threadsPerBlock.x),
                                 getNumberCudaBlocks(targetHeight, threadsPerBlock.y)};
            const auto& sourceSize = sourceSizes[0];
            const auto sourceHeight = sourceSize[2];
            const auto sourceWidth = sourceSize[3];

            // No multi-scale merging or no merging required
            if (sourceSizes.size() == 1)
            {
                const auto num = sourceSize[0];
                if (targetSize[0] > 1 || num == 1)
                {
const auto REPS = 1;
// const auto REPS = 1;
double timeNormalize0 = 0.;
double timeNormalize1 = 0.;
double timeNormalize2 = 0.;
double timeNormalize3 = 0.;
double timeNormalize4 = 0.;
double timeNormalize5 = 0.;
// OP_CUDA_PROFILE_INIT(5);
//                     // Option a)
//                     const auto sourceChannelOffset = sourceHeight * sourceWidth;
//                     const auto targetChannelOffset = targetWidth * targetHeight;
//                     for (auto n = 0; n < num; n++)
//                     {
//                         const auto offsetBase = n*channels;
//                         for (auto c = 0 ; c < channels ; c++)
//                         {
//                             const auto offset = offsetBase + c;
//                             resizeKernel<<<numBlocks, threadsPerBlock>>>(targetPtr + offset * targetChannelOffset,
//                                                                          sourcePtrs.at(0) + offset * sourceChannelOffset,
//                                                                          sourceWidth, sourceHeight, targetWidth,
//                                                                          targetHeight);
//                         }
//                     }
// OP_CUDA_PROFILE_END(timeNormalize0, 1e3, 5);
// OP_CUDA_PROFILE_INIT(REPS);
//                     // Option a)
//                     const auto sourceChannelOffset = sourceHeight * sourceWidth;
//                     const auto targetChannelOffset = targetWidth * targetHeight;
//                     for (auto n = 0; n < num; n++)
//                     {
//                         const auto offsetBase = n*channels;
//                         for (auto c = 0 ; c < channels ; c++)
//                         {
//                             const auto offset = offsetBase + c;
//                             resizeKernel<<<numBlocks, threadsPerBlock>>>(targetPtr + offset * targetChannelOffset,
//                                                                          sourcePtrs.at(0) + offset * sourceChannelOffset,
//                                                                          sourceWidth, sourceHeight, targetWidth,
//                                                                          targetHeight);
//                         }
//                     }
// OP_CUDA_PROFILE_END(timeNormalize1, 1e3, REPS);
// OP_CUDA_PROFILE_INIT(REPS);
//                     // Option a)
//                     const dim3 threadsPerBlock{512, 1};
//                     const dim3 numBlocks{getNumberCudaBlocks(targetWidth, threadsPerBlock.x),
//                                          getNumberCudaBlocks(targetHeight, threadsPerBlock.y)};
//                     const auto sourceChannelOffset = sourceHeight * sourceWidth;
//                     const auto targetChannelOffset = targetWidth * targetHeight;
//                     for (auto n = 0; n < num; n++)
//                     {
//                         const auto offsetBase = n*channels;
//                         for (auto c = 0 ; c < channels ; c++)
//                         {
//                             const auto offset = offsetBase + c;
//                             resizeKernel<<<numBlocks, threadsPerBlock>>>(targetPtr + offset * targetChannelOffset,
//                                                                          sourcePtrs.at(0) + offset * sourceChannelOffset,
//                                                                          sourceWidth, sourceHeight, targetWidth,
//                                                                          targetHeight);
//                         }
//                     }
// OP_CUDA_PROFILE_END(timeNormalize2, 1e3, REPS);
// OP_CUDA_PROFILE_INIT(REPS);
//                     // Option b)
//                     const dim3 threadsPerBlock{512, 1, 1};
//                     const dim3 numBlocks{getNumberCudaBlocks(targetWidth, threadsPerBlock.x),
//                                          getNumberCudaBlocks(targetHeight, threadsPerBlock.y),
//                                          getNumberCudaBlocks(num * channels, threadsPerBlock.z)};
//                     resizeAllKernel<<<numBlocks, threadsPerBlock>>>(
//                         targetPtr, sourcePtrs.at(0), sourceWidth, sourceHeight, targetWidth, targetHeight,
//                         num * channels);
// OP_CUDA_PROFILE_END(timeNormalize3, 1e3, REPS);
// OP_CUDA_PROFILE_INIT(REPS);
//                     // Option b)
//                     const dim3 threadsPerBlock{THREADS_PER_BLOCK_1D, THREADS_PER_BLOCK_1D, 1};
//                     const dim3 numBlocks{getNumberCudaBlocks(targetWidth, threadsPerBlock.x),
//                                          getNumberCudaBlocks(targetHeight, threadsPerBlock.y),
//                                          getNumberCudaBlocks(num * channels, threadsPerBlock.z)};
//                     resizeAllKernel<<<numBlocks, threadsPerBlock>>>(
//                         targetPtr, sourcePtrs.at(0), sourceWidth, sourceHeight, targetWidth, targetHeight,
//                         num * channels);
// OP_CUDA_PROFILE_END(timeNormalize4, 1e3, REPS);
OP_CUDA_PROFILE_INIT(REPS);
                    // Option b)
                    const auto rescaleFactor = (unsigned int) std::ceil((float)(targetHeight) / (float)(sourceHeight));

                    const dim3 threadsPerBlock{rescaleFactor, rescaleFactor, 1};
                    const dim3 numBlocks{getNumberCudaBlocks(targetWidth, threadsPerBlock.x),
                                         getNumberCudaBlocks(targetHeight, threadsPerBlock.y),
                                         getNumberCudaBlocks(num * channels, threadsPerBlock.z)};
                    resizeAllKernelShared<<<numBlocks, threadsPerBlock>>>(
                        targetPtr, sourcePtrs.at(0), sourceWidth, sourceHeight, targetWidth, targetHeight,
                        num * channels, rescaleFactor);
OP_CUDA_PROFILE_END(timeNormalize5, 1e3, REPS);
log("  Res1(ori)=" + std::to_string(timeNormalize1) + "ms");
log("  Res2(ori)=" + std::to_string(timeNormalize2) + "ms");
log("  Res3(new)=" + std::to_string(timeNormalize3) + "ms");
log("  Res4(new)=" + std::to_string(timeNormalize4) + "ms");
log("  Res5(new)=" + std::to_string(timeNormalize5) + "ms");
                }
                // Old inefficient multi-scale merging
                else
                    error("It should never reache this point. Notify us otherwise.", __LINE__, __FUNCTION__, __FILE__);
            }
            // Multi-scaling merging
            else
            {
                const auto targetChannelOffset = targetWidth * targetHeight;
                hipMemset(targetPtr, 0, channels*targetChannelOffset * sizeof(T));
                const auto scaleToMainScaleWidth = targetWidth / T(sourceWidth);
                const auto scaleToMainScaleHeight = targetHeight / T(sourceHeight);

                for (auto i = 0u ; i < sourceSizes.size(); i++)
                {
                    const auto& currentSize = sourceSizes.at(i);
                    const auto currentHeight = currentSize[2];
                    const auto currentWidth = currentSize[3];
                    const auto sourceChannelOffset = currentHeight * currentWidth;
                    const auto scaleInputToNet = scaleInputToNetInputs[i] / scaleInputToNetInputs[0];
                    const auto scaleWidth = scaleToMainScaleWidth / scaleInputToNet;
                    const auto scaleHeight = scaleToMainScaleHeight / scaleInputToNet;
                    // All but last image --> add
                    if (i < sourceSizes.size() - 1)
                    {
                        for (auto c = 0 ; c < channels ; c++)
                        {
                            resizeKernelAndAdd<<<numBlocks, threadsPerBlock>>>(
                                targetPtr + c * targetChannelOffset, sourcePtrs[i] + c * sourceChannelOffset,
                                scaleWidth, scaleHeight, currentWidth, currentHeight, targetWidth,
                                targetHeight
                            );
                        }
                    }
                    // Last image --> average all
                    else
                    {
                        for (auto c = 0 ; c < channels ; c++)
                        {
                            resizeKernelAndAverage<<<numBlocks, threadsPerBlock>>>(
                                targetPtr + c * targetChannelOffset, sourcePtrs[i] + c * sourceChannelOffset,
                                scaleWidth, scaleHeight, currentWidth, currentHeight, targetWidth,
                                targetHeight, (int)sourceSizes.size()
                            );
                        }
                    }
                }
            }

            cudaCheck(__LINE__, __FUNCTION__, __FILE__);
        }
        catch (const std::exception& e)
        {
            error(e.what(), __LINE__, __FUNCTION__, __FILE__);
        }
    }

    template void resizeAndMergeGpu(
        float* targetPtr, const std::vector<const float*>& sourcePtrs, const std::array<int, 4>& targetSize,
        const std::vector<std::array<int, 4>>& sourceSizes, const std::vector<float>& scaleInputToNetInputs);
    template void resizeAndMergeGpu(
        double* targetPtr, const std::vector<const double*>& sourcePtrs, const std::array<int, 4>& targetSize,
        const std::vector<std::array<int, 4>>& sourceSizes, const std::vector<double>& scaleInputToNetInputs);
}
